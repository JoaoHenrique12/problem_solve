
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void cuda_hello() {
  printf("Na GPU!\n");
}

int main(int argc, char *argv[])
{
  printf("Na CPU!\n");
  cuda_hello<<<1,1>>>();
  hipDeviceSynchronize();
  return EXIT_SUCCESS;
}
