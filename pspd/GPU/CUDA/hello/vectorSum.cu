
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void add(int N, float* a, float* b, float* c) {
  int tid = threadIdx.x;
  if (tid < N)
    c[tid] = a [tid] + b[tid];
}

int main(int argc, char *argv[])
{
  float *a, *b, *c;
  int N = 1000;
  a = (float*) malloc(N*sizeof(float));
  b = (float*) malloc(N*sizeof(float));
  c = (float*) malloc(N*sizeof(float));

  for (int i = 0; i < N; i++)
    { a[i] = -i; b[i] = i * i; }

  float *dev_a, *dev_b, *dev_c;
  hipMalloc((void**)&dev_a, N*sizeof(float));
  hipMalloc((void**)&dev_b, N*sizeof(float));
  hipMalloc((void**)&dev_c, N*sizeof(float));

  hipMemcpy(dev_a, a, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, N*sizeof(float), hipMemcpyHostToDevice);
  add<<<1,N>>>(N, dev_a, dev_b, dev_c);
  hipMemcpy(dev_c, c, N*sizeof(float), hipMemcpyDeviceToHost);

  free(a); free(b); free(c);
  hipFree(dev_a); hipFree(dev_b); hipFree(dev_c);
  return 0;
}
