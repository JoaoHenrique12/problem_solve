
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void add(int a, int b, int* c) {
  *c = a + b;
}

int main(int argc, char *argv[])
{
  int a = 10, b = 30, c = -1;
  int *dev_c;
  
  // Allocate memory on the GPU
  hipMalloc((void**)&dev_c, sizeof(int));
  
  // Launch the kernel
  add<<<1,1>>>(a, b, dev_c);
  
  // Wait for kernel to finish
  hipDeviceSynchronize();
  
  // Copy result back to host
  hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
  
  printf("%d + %d = %d\n", a, b, c);

  // Free GPU memory
  hipFree(dev_c);
  
  return EXIT_SUCCESS;
}
